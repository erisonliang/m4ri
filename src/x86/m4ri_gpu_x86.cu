
#include <hip/hip_runtime.h>
#include <stdint.h>

namespace x86 
{
	namespace gpu
	{

		#define UINT32_BIT_SIZE 32

		__device__ 
		bool inline get_bit(const uint32_t* arr, uint32_t n)
		{
			uint32_t cell_idx = n / UINT32_BIT_SIZE;
			return arr[cell_idx] & (1 << (n % UINT32_BIT_SIZE));
		}


		__device__ 
		void inline set_bit(uint32_t* arr, uint32_t n, uint32_t bit_value)
		{
			uint32_t cell_idx = n / UINT32_BIT_SIZE;
			if(bit_value)
				arr[cell_idx] = arr[cell_idx] | (1 << (n % UINT32_BIT_SIZE));
			else
				arr[cell_idx] = arr[cell_idx] & ~(1 << (n % UINT32_BIT_SIZE));
		}

		__global__ 
		void m4ri_multiply(
			const uint32_t* A, uint32_t a_rows, uint32_t a_cols,
			const uint32_t* B, uint32_t b_rows, uint32_t b_cols, 
			uint32_t* C, const bool* precalc_matrix, unsigned int k) 
		{
			// index - номер строки в матрице А и В
			unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;

			if(index < a_rows)
			{
				for (unsigned int i = 0; i < b_rows; i++)											 
				{
					bool sum = 0;
					for(unsigned int j = 0; j < b_cols / k; j++)
					{
						// получаем нужный uint32_t элемент в массиве A и B
						unsigned int id_A = (index * a_cols + j * k) / UINT32_BIT_SIZE;
						unsigned int id_B = (i * b_cols + j * k) / UINT32_BIT_SIZE;

						// получаем смещение (должно быть кратно k)
						unsigned int offset_A = (index * a_cols + j * k) % UINT32_BIT_SIZE;
						unsigned int offset_B = (i * b_cols + j * k) % UINT32_BIT_SIZE;

						uint32_t k_vector_A = (A[id_A] >> offset_A) & ((1 << k) - 1);
						uint32_t k_vector_B = (B[id_B] >> offset_B) & ((1 << k) - 1);

						sum ^= precalc_matrix[k_vector_A * (1 << k) + k_vector_B];
					}
					set_bit(C, index * b_rows + i, sum);
				}
			}
		}

		__global__
		void transpose(uint32_t* dst, const uint32_t* src, uint32_t rows, uint32_t cols)
		{
			for(int i = 0; i < rows; ++i)
				for (int j = 0; j < cols; ++j)
					set_bit(dst, i * cols + j, get_bit(src, j * rows + i));
		}


		__global__
		void m4ri_precalc(bool* precalc_matrix, uint32_t bits)
		{
			for(unsigned int i = 0; i < (1 << bits); i++)
			{
				for(unsigned int j = 0; j < (1 << bits); j++)
				{
					bool scalar_product = 0;
					for(unsigned int k = 0; k < bits; k++)
					{
						scalar_product ^= get_bit(&i, k) & get_bit(&j, k);
					}
					precalc_matrix[i * (1 << bits) + j] = scalar_product;
				}
			}
		}

		__global__
		void mar_multiply(const uint32_t* A, uint32_t a_rows, uint32_t a_cols,
						  const uint32_t* B, uint32_t b_rows, uint32_t b_cols, uint32_t* C)
		{
			unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;

			for(unsigned int i = 0; i < a_cols; i++)
			{
				bool bit = get_bit(A, index * a_cols + i);
				for(unsigned int j = 0; j < b_cols / UINT32_BIT_SIZE; j++)
				{
					C[index * a_cols / UINT32_BIT_SIZE + j] ^= B[i * b_cols / UINT32_BIT_SIZE + j] * bit;
				}
			}
		}
	}
}